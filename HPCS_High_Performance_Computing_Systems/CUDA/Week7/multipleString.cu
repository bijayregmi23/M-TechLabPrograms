#include "hip/hip_runtime.h"

#include<stdio.h>
#include<string.h>

__global__ void multipleString(char* a,char* b,int n,int dup){
    int i = threadIdx.x;
    for(int j=0;j<dup;j++){
        b[i+j*n] = a[i];
    }
}

int main(void){
    
    char *d_a, *d_b;
    int n, dup;
    char a[100], b[100];
    
    printf("\n\nEnter string : ");
    scanf("%s",a);
    n = strlen(a);
     
    printf("\nEnter number of Duplication : ");
    scanf("%d",&dup);
    
    hipMalloc((void**)&d_a, ((sizeof(char))*n*dup));
    hipMalloc((void**)&d_b, ((sizeof(char))*n*dup));
    
    hipMemcpy(d_a,&a,((sizeof(char))*n), hipMemcpyHostToDevice);
    multipleString<<<1,n>>>(d_a, d_b, n, dup);
    hipMemcpy(&b,d_b,((sizeof(char))*n*dup),hipMemcpyDeviceToHost);

    printf("\n\nResultant String : ");
    for (int i = 0; i <(n*dup); i++)
        printf("%c",b[i]);

    printf("\n\n");
    hipFree(d_a);
    hipFree(d_b);
    return 0;    
}