#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void testKernel(int *in, int *out, int size)
{
    bool oddeven = true;
    __shared__ bool swappedodd;
    __shared__ bool swappedeven;
    int temp;
    while (1)
    {
        if (oddeven == true)
        {
            __syncthreads();
            swappedeven = false;
            __syncthreads();
            if (threadIdx.y == 0)
            {
                int idx = threadIdx.x;
                if (idx < (size / 2))
                {
                    if (in[2 * idx] > in[2 * idx + 1])
                    {
                        temp = in[2 * idx];
                        in[2 * idx] = in[2 * idx + 1];
                        in[2 * idx + 1] = temp;
                        swappedeven = true;
                    }
                }
            }
            __syncthreads();
        }
        else
        {
            __syncthreads();
            swappedodd = false;
            __syncthreads();
            if (threadIdx.y == 0)
            {
                int idx = threadIdx.x;
                if (idx < (size / 2) - 1)
                {
                    if (in[2 * idx + 1] > in[2 * idx + 2])
                    {

                        temp = in[2 * idx + 1];
                        in[2 * idx + 1] = in[2 * idx + 2];
                        in[2 * idx + 2] = temp;
                        swappedodd = true;
                    }
                }
            }
            __syncthreads();
        }
        if (!(swappedodd || swappedeven))
            break;
        oddeven = !oddeven; // switch phase of sorting
    }
    __syncthreads();
    // Store this phase's in[] array to out[] array
    int idx = threadIdx.x;
    if (idx < size)
        out[idx] = in[idx];
}

int main(void)
{
    int i;
    int *a_sorted;
    int *d_a, *d_sorted;
    int n = 6; // make sure to keep this even
    int size = sizeof(int) * n;
    hipMalloc((void **)&d_a, size);
    hipMalloc((void **)&d_sorted, size);
    printf("\nEnter Size : ");
    scanf("%d",&n);
    int a[6] = {99, 4, 2, 87, 54, 1};
    a_sorted = (int *)malloc(size);
    printf("\nEnter Elements of Array : ");
    for (i = 0; i < n; i++)
    {
        printf("%d", a[i]);
    }
    printf("\n");
    // d_a -> destination. a -> source.
    // Host to device array copy
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    //<<< >>> CUDA semantic
    testKernel<<<1, n> > >(d_a, d_sorted, n);
    // Device to Host array for final display (I/O)
    hipMemcpy(a_sorted, d_sorted, size, hipMemcpyDeviceToHost);
    printf("\n\nSorted Array  : ");
    for (i = 0; i < n; i++)
    {
        printf("%d\t", a_sorted[i]);
    }
    printf("\n\n");
    // free memory allocated by malloc and cudamalloc
    free(a_sorted);
    hipFree(d_sorted);
    hipFree(d_a);
}
