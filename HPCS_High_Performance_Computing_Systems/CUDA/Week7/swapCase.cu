#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void swapCase(char *a){
    int i = threadIdx.x;
    if (a[i] >= 'a' && a[i] <= 'z')
        a[i] = a[i] - 32;
    else if (a[i] >= 'A' && a[i] <= 'Z')
        a[i] = a[i] + 32;
}

int main(void)
{

    char *d_a;
    int n=0;

    char a[100];

    printf("\nEnter String : ");
    gets(a);

    while( a[n] != '\0')
        n++ ;
    n += 1;

    hipMalloc((void **)&d_a, (sizeof(char) * n));

    
    hipMemcpy(d_a, &a, (sizeof(char) * n), hipMemcpyHostToDevice);
    
    swapCase<<<1,256>>>(d_a);

    hipMemcpy(&a, d_a, (sizeof(char) * n), hipMemcpyDeviceToHost);

    printf("\nResultant String : %s\n\n",a);

    hipFree(d_a);
    // hipFree(d_b);

    return 0;
}