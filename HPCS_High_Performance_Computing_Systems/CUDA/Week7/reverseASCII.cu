#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void characterReverse(char *a, char *b, int n){
    int i = threadIdx.x;
    if(i == n-1)
        b[i] = a[i];
    else{
        int value = (int)a[i];
        int r = 0;
        while (value > 0) {
            int digit = value % 10;
            r = r * 10 + digit;
            value /= 10;
        }
        b[i] = (char)r;
    }
}

int main(void)
{

    char *d_a, *d_b;
    int n=0;

    char a[100], b[100];

    printf("\nEnter String : ");
    gets(a);

    while( a[n] != '\0')
        n++ ;
    n += 1;

    hipMalloc((void **)&d_a, (sizeof(char) * n));
    hipMalloc((void **)&d_b, (sizeof(char) * n));
    
    hipMemcpy(d_a, &a, (sizeof(char) * n), hipMemcpyHostToDevice);
    characterReverse<<<1,256>>>(d_a, d_b, n);
    hipMemcpy(&b, d_b, (sizeof(char) * n), hipMemcpyDeviceToHost);


    printf("\nResultant String : %s\n\n",b);

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}