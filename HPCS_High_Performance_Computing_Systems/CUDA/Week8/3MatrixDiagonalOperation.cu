#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

__global__ void gpu_diagonal_op(int *a, int n)
{
    int idx = blockIdx.x;
    int idy = blockIdx.y;

    int element_replace;
    if (idx == idy)
    {
        element_replace = 0;
    }
    else if (idx > idy)
    {
        int fact = 1;
        int element = a[idy * n + idx];
        for (int i = 1; i <= element; i++)
        {
            fact = fact * i;
        }
        element_replace = fact;
    }
    else
    {
        int sum = 0, rem;
        int element = a[idy * n + idx];
        while (element > 0)
        {
            rem = element % 10;
            sum = sum + rem;
            element = element / 10;
        }
        element_replace = sum;
    }
    a[idy * n + idx] = element_replace;
}

int main(int argc, char const *argv[])
{
    int n;
    printf("\nEnter Size : ");
    scanf("%d", &n);
    // allocate memory in host RAM
    int *h_a;
    hipHostMalloc((void **)&h_a, sizeof(int) * n * n);
    // random initialize matrix A
    printf("\nEnter Matrix A : \n");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            scanf("%d", &h_a[i * n + j]);
        }
    }

    // Allocate memory space on the device
    int *d_a;
    hipMalloc((void **)&d_a, sizeof(int) * n * n);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int) * n * n, hipMemcpyHostToDevice);

    dim3 dimGrid(n, n);
    dim3 dimBlock(1, 1);

    gpu_diagonal_op<<<dimGrid, dimBlock> > >(d_a, n);

    // Transfer results from device to host
    hipMemcpy(h_a, d_a, sizeof(int) * n * n, hipMemcpyDeviceToHost);

    // validate results computed by GPU
    printf("\n\nResultant Matrix : \n");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", h_a[i * n + j]);
        }
        printf("\n");
    }

    // free memory
    hipFree(d_a);

    hipHostFree(h_a);

    return 0;
}
