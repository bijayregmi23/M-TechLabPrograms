#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void octalConversion(int *a, int *b){
    int i = threadIdx.x;

    // Converts decimal to octal
    int n = a[i], res = 0, multiplier = 1;
    while(n!=0){
        res = res + (multiplier * (n%8));
        n = n/8;
        multiplier = multiplier * 10;
    }

    b[i] = res;
}

int main(void){
    int *d_a, *d_b, n;
    
    printf("\nEnter size of array : ");
    scanf("%d", &n);

    int a[n], b[n], i;
    hipMalloc((void **)&d_a, (sizeof(int) * n));
    hipMalloc((void **)&d_b, (sizeof(int) * n));

    
    printf("\nEnter the elements of array : ");
    for (i = 0; i < n; i++)
        scanf("%d", &a[i]);
    printf("\n");
    
    hipMemcpy(d_a, &a, (sizeof(int) * n), hipMemcpyHostToDevice);
    
    octalConversion<<<1,256>>>(d_a, d_b);

    hipMemcpy(&b, d_b, (sizeof(int) * n), hipMemcpyDeviceToHost);
    printf("\nResultant octal values : ");
    for (int i = 0; i < n; i++)
        printf("%d  ", b[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}