#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m)
    {
        for (int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

int main(int argc, char const *argv[])
{
    int m, n, k;

    printf("\nEnter M : ");
    scanf("%d", &m);
    printf("\nEnter N : ");
    scanf("%d", &n);
    printf("\nEnter K : ");
    scanf("%d", &k);

    // allocate memory in host RAM
    int *h_a, *h_b, *h_c;
    hipHostMalloc((void **)&h_a, sizeof(int) * m * n);
    hipHostMalloc((void **)&h_b, sizeof(int) * n * k);
    hipHostMalloc((void **)&h_c, sizeof(int) * m * k);

    // random initialize matrix A
    printf("\n\nEnter Matrix A : \n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            scanf("%d", &h_a[i * n + j]);
        }
    }

    // random initialize matrix B
    printf("\nEnter Matrix B : \n");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            scanf("%d", &h_b[i * k + j]);
        }
    }

    float gpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    // Allocate memory space on the device
    int *d_a, *d_b, *d_c;
    hipMalloc((void **)&d_a, sizeof(int) * m * n);
    hipMalloc((void **)&d_b, sizeof(int) * n * k);
    hipMalloc((void **)&d_c, sizeof(int) * m * k);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int) * m * n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int) * n * k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    gpu_matrix_mult<<<dimGrid, dimBlock> > >(d_a, d_b, d_c, m, n, k);

    // Transfer results from device to host
    hipMemcpy(h_c, d_c, sizeof(int) * m * k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("\nTime elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);

    // validate results computed by GPU
    printf("\n\nResultant Matrix C : \n");
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < k; ++j)
        {
            printf("%d ", h_c[i * k + j]);
        }
        printf("\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}
