#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void addArray(int *a, int *b, int *c){
    int j = threadIdx.x;
    c[j] = a[j] + b[j];
}

int main(void)
{
    int n;
    //Defining the size of an array
    printf("\nEnter size of array : ");
    scanf("%d", &n);

    // Initialising variables of the Host
    int a[n * 2], b[n * 2], c[n * 2], i;

    // Taking input of all A and B in Host Variables
    printf("\nEnter elements of A : ");
    for (i = 0; i < n; i++)
        scanf("%d", &a[i]);
    printf("\nEnter elements of B : ");
    for (i = 0; i < n; i++)
        scanf("%d", &b[i]);

    // Initialising variables for Device
    int *d_a, *d_b, *d_c;

    // Allocating Memory for all Device Variables
    hipError_t cudaStatus = hipMalloc((void **)&d_a, (sizeof(int) * n));
    if (cudaStatus != hipSuccess){
        printf("CUDA Error1: %s\n", hipGetErrorString(cudaStatus));
    }
    hipMalloc((void **)&d_b, (sizeof(int) * n));
    hipMalloc((void **)&d_c, (sizeof(int) * n));

    // Coping values of all the Host variables to Device variables
    hipMemcpy(d_a, &a, (sizeof(int) * n), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, (sizeof(int) * n), hipMemcpyHostToDevice);
    
    // Calling kernal to add Device variables A and B, and to store it in C
    addArray<<<1, n>>>(d_a, d_b, d_c);

    // Copying Device Variables to the Host variable
    hipMemcpy(&c, d_c, (sizeof(int) * n), hipMemcpyDeviceToHost);

    // Printing value stored in Host variable
    printf("\nResultant Array : ");
    for (i = 0; i < n; i++)
        printf("%d\t",c[i]);
    printf("\n\n");

    // Deallocating all the Host variables
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}