#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>

__global__ void characterReverse(char *a, char *b, int n){
    int i = threadIdx.x;
    if(i == n-1)
        b[i] = a[i];
    else
        b[i] = a[n-i-2];
}

int main(void)
{

    char *d_a, *d_b;
    int n=0;

    char a[100], b[100];

    printf("\nEnter String : ");
    gets(a);

    while( a[n] != '\0')
        n++ ;
    n += 1;

    hipMalloc((void **)&d_a, (sizeof(char) * n));
    hipMalloc((void **)&d_b, (sizeof(char) * n));
    
    hipMemcpy(d_a, &a, (sizeof(char) * n), hipMemcpyHostToDevice);
    characterReverse<<<1,256>>>(d_a, d_b, n);
    hipMemcpy(&b, d_b, (sizeof(char) * n), hipMemcpyDeviceToHost);


    printf("\nResultant String : %s\n\n",b);

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}