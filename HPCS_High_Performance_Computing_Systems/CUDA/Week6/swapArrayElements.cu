#include "hip/hip_runtime.h"

#include <stdio.h>

__global__ void swapAlternate(int *a, int *b, int n){
    int i = threadIdx.x;
    if (i % 2 == 0){
        b[i] = a[i + 1];
        b[i + 1] = a[i];
    }
    if(n%2 == 1)
        b[n-1] = a[n-1];
}

int main(void){
    int *d_a, *d_b, n;
    
    printf("\nEnter size of array : ");
    scanf("%d", &n);

    int a[n], b[n], i;
    hipMalloc((void **)&d_a, (sizeof(int) * n));
    hipMalloc((void **)&d_b, (sizeof(int) * n));

    
    printf("\nEnter the elements of array : ");
    for (i = 0; i < n; i++)
        scanf("%d", &a[i]);
    printf("\n");
    
    hipMemcpy(d_a, &a, (sizeof(int) * n), hipMemcpyHostToDevice);
    
    swapAlternate<<<1,256>>>(d_a, d_b, n);

    hipMemcpy(&b, d_b, (sizeof(int) * n), hipMemcpyDeviceToHost);
    printf("\nResultant array : ");
    for (int i = 0; i < n; i++)
        printf("%d  ", b[i]);
    printf("\n");

    hipFree(d_a);
    hipFree(d_b);
    return 0;
}