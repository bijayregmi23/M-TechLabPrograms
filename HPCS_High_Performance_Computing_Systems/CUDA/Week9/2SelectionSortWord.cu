#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <string.h>

__global__ void sort(char *in, char *out, int n)
{
    int i = threadIdx.x; // current thread
    char iData = in[i];
    char iKey = iData;
    // Compute position of in[i] in output
    int pos = 0;
    for (int j = 0; j < n; j++)
    {
        char jKey = in[j]; // broadcasted
        int m = (jKey < iKey) || (jKey == iKey && j < i);
        pos += (m) ? 1 : 0;
    }
    out[pos] = iData;
}
int main()
{
    int n;
    char b[100];

    printf("\nEnter String : ");
    char string[8] = "HPCSLAB";
    n = strlen(string);
    printf("\nORIGINAL String : ");
    printf("%s", string);
    char *d_data_sorted, *d_data;
    int *count;
    hipMalloc((void **)&d_data, sizeof(char) * n);
    hipMalloc((void **)&d_data_sorted, sizeof(char) * n);
    hipMalloc((void **)&count, sizeof(int));
    hipMemcpy(d_data, &string, sizeof(char) * n, hipMemcpyHostToDevice);
    hipMemcpy(count, &n, sizeof(int), hipMemcpyHostToDevice);
    sort<<<1, n> > >(d_data, d_data_sorted, n);
    hipMemcpy(b, d_data_sorted, sizeof(char) * n, hipMemcpyDeviceToHost);
    printf("\n\nSORTED String : ");
    printf("%s", b);
    printf("\n");
}
