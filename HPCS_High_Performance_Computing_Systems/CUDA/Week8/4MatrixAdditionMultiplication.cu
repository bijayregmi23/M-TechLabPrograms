#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult2d(int *a,int *b, int *c,int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < n && row < n)
    {
        for(int i = 0; i < n; i++)
        {
            sum += a[row * n + i] * b[i * n + col];
        }
        c[row * n + col] = sum;
    }
}

__global__ void gpu_matrix_add2d(int *a,int *b, int *d, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    d[row*blockDim.x+col]=a[row*blockDim.x+col]+b[row*blockDim.x+col];
}


int main(int argc, char const *argv[])
{
    int n;
    printf("\nEnter Size : ");
    scanf("%d",&n);

    // allocate memory in host RAM
    int *h_a, *h_b, *h_c, *h_d;
    hipHostMalloc((void **) &h_a, sizeof(int)*n*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*n);
    hipHostMalloc((void **) &h_c, sizeof(int)*n*n);
hipHostMalloc((void **) &h_d, sizeof(int)*n*n);

    // random initialize matrix A
    printf("\nEnter Matrix A : \n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
        	scanf("%d",&h_a[i * n + j]);
        }
    }

    // random initialize matrix B
    printf("\nEnter Matrix B : \n");
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < n; ++j) {
        	scanf("%d",&h_b[i * n + j]);
        }
    }

    float gpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    // Allocate memory space on the device
    int *d_a, *d_b, *d_c , *d_d;
    hipMalloc((void **) &d_a, sizeof(int)*n*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*n);
    hipMalloc((void **) &d_c, sizeof(int)*n*n);
 hipMalloc((void **) &d_d, sizeof(int)*n*n);

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*n*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*n, hipMemcpyHostToDevice);

    unsigned int grid_rows = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (n + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
   
    gpu_matrix_mult2d<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);  
 gpu_matrix_add2d<<<dimGrid, dimBlock>>>(d_a, d_b, d_d,n);
  
    
    // Transfer results from device to host
    hipMemcpy(h_c, d_c, sizeof(int)*n*n, hipMemcpyDeviceToHost);
hipMemcpy(h_d, d_d, sizeof(int)*n*n, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("\nTime elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", n, n, n, n, gpu_elapsed_time_ms);


    // validate results computed by GPU
    printf("\n\nResultant Matrix C :\n");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", h_c[i*n + j]);
        }
        printf("\n");
    }


printf("\nResultant Matrix D :\n");
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            printf("%d ", h_d[i*n + j]);
        }
        printf("\n");
    }

    // free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    return 0;
}
